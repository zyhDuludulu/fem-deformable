#include "hip/hip_runtime.h"
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <glm/vec2.hpp>
#include <cstdio>
#include <string>
#include "fem/parameters.h"
#include "fem/simulator.h"
#include "fem/solver.h"
#include "glm/ext/matrix_float2x2.hpp"
#include "glm/ext/matrix_transform.hpp"
#include "glm/ext/vector_float2.hpp"
#include "glm/matrix.hpp"

struct Constants {
  float dt;
  float g;
  float mu;
  float lambda;

  glm::vec2 *position;
  glm::vec2 *velocity;
  glm::vec2 *acceleration;
  glm::ivec3 *triangles;
  float *A;
  glm::mat2 *DmInv;
};

__constant__ Constants cuConstants;

/* The data in the simulator must have been initialized, 
   i.e. createMesh() before setUp() */
void Solver::setUp(Simulator *sim) {
  this->sim = sim;
  int device_count = 0;
  std::string name;
  hipError_t err = hipGetDeviceCount(&device_count);

  printf("---------------------------------------------------------\n");
  printf("Initializing CUDA for CudaRenderer\n");
  printf("Found %d CUDA devices\n", device_count);

  for (int i = 0; i < device_count; i++) {
    hipDeviceProp_t deviceProps;
    hipGetDeviceProperties(&deviceProps, i);
    name = deviceProps.name;
    printf("Device %d: %s\n", i, deviceProps.name);
    printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
    printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
    printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
  }
  printf("---------------------------------------------------------\n");

  hipMalloc(&cudaDevicePosition,     N_POINTS * sizeof(glm::vec2));
  hipMalloc(&cudaDeviceVelocity,     N_POINTS * sizeof(glm::vec2));
  hipMalloc(&cudaDeviceAcceleration, N_POINTS * sizeof(glm::vec2));
  hipMalloc(&cudaDeviceTriangles,    N_TRIANGLES * sizeof(glm::ivec3));
  cp2GPU();

  hipMalloc(&cudaDeviceA,            N_TRIANGLES * sizeof(glm::mat2));
  hipMalloc(&cudaDeviceDmInv,        N_TRIANGLES * sizeof(glm::mat2));

  Constants constants;
  constants.dt = params::dt;
  constants.g = params::g;
  constants.mu = params::mu;
  constants.lambda = params::lambda;
  constants.position = sim->x;
  constants.velocity = sim->v;
  constants.acceleration = sim->f;
  constants.triangles = sim->triangles;
  constants.A = sim->A;

  hipMemcpyToSymbol(HIP_SYMBOL(cuConstants), &constants, sizeof(Constants));

}

void Solver::cp2GPU() {
  hipMemcpy(cudaDevicePosition,      sim->x,     N_POINTS * sizeof(glm::vec2), hipMemcpyHostToDevice);
  hipMemcpy(cudaDeviceVelocity,      sim->v,     N_POINTS * sizeof(glm::vec2), hipMemcpyHostToDevice);
  hipMemcpy(cudaDeviceAcceleration,  sim->f,     N_POINTS * sizeof(glm::vec2), hipMemcpyHostToDevice);
  hipMemcpy(cudaDeviceTriangles, sim->triangles, N_TRIANGLES * sizeof(glm::ivec3), hipMemcpyHostToDevice);
}

void Solver::cp2CPU() {
  hipMemcpy(sim->x, cudaDevicePosition, N_POINTS, hipMemcpyDeviceToHost);
}

__global__
void kernelComputeDmInv() {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int a = cuConstants.triangles[i].x;
  int b = cuConstants.triangles[i].y;
  int c = cuConstants.triangles[i].z;
  glm::mat2 Dm = glm::mat2(cuConstants.position[b] - cuConstants.position[a], cuConstants.position[c] - cuConstants.position[a]);
  cuConstants.DmInv[i] = glm::inverse(Dm);
  cuConstants.A[i] = 0.5f * glm::determinant(Dm);
}

void Solver::computeDmInv() {
  int threadsPerBlock = 256;
  int blocksPerGrid = (N_TRIANGLES + threadsPerBlock - 1) / threadsPerBlock;
  kernelComputeDmInv<<<blocksPerGrid, threadsPerBlock>>>();
  hipDeviceSynchronize();
}

__global__
void kernelInitAcceleration() {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  cuConstants.acceleration[i] = glm::vec2(0.0f, 0.0f);
}

__global__
void kernelComputeForces() {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int a = cuConstants.triangles[i].x;
  int b = cuConstants.triangles[i].y;
  int c = cuConstants.triangles[i].z;
  glm::vec2 x_a = cuConstants.position[a];
  glm::vec2 x_b = cuConstants.position[b];
  glm::vec2 x_c = cuConstants.position[c];
  glm::mat2 Ds = glm::mat2(x_b - x_a, x_c - x_a);
  glm::mat2 F = Ds * cuConstants.DmInv[i];
  glm::mat2 E = 0.5f * (glm::transpose(F) * F - glm::identity<glm::mat2>());
  glm::mat2 P = F * (2.0f * cuConstants.mu * E);
  glm::mat2 grad = glm::transpose(cuConstants.A[i] * P * glm::transpose(cuConstants.DmInv[i]));
  atomicAdd(&cuConstants.acceleration[a].x, -grad[0][0]);
  atomicAdd(&cuConstants.acceleration[a].y, -grad[0][1]);
  atomicAdd(&cuConstants.acceleration[b].x, -grad[1][0]);
  atomicAdd(&cuConstants.acceleration[b].y, -grad[1][1]);
  atomicAdd(&cuConstants.acceleration[c].x, -grad[2][0]);
  atomicAdd(&cuConstants.acceleration[c].y, -grad[2][1]);
}

__global__
void kernelUpdatePosition() {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  cuConstants.velocity[i] += cuConstants.acceleration[i] * cuConstants.dt;
  cuConstants.position[i] += cuConstants.velocity[i] * cuConstants.dt;
}

void Solver::solve() {
  for (int i = 0; i < params::sub_steps; i++) {
    kernelInitAcceleration<<<N_POINTS, 1>>>();
    hipDeviceSynchronize();
    kernelComputeForces<<<N_TRIANGLES, 1>>>();
    hipDeviceSynchronize();
    kernelUpdatePosition<<<N_POINTS, 1>>>();
  }
  cp2CPU();
}

