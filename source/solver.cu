#include "hip/hip_runtime.h"
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <glm/vec2.hpp>
#include <cstdio>
#include <string>
#include "fem/parameters.h"
#include "fem/simulator.h"
#include "fem/solver.h"
#include "glm/exponential.hpp"
#include "glm/ext/matrix_float2x2.hpp"
#include "glm/ext/matrix_transform.hpp"
#include "glm/ext/vector_float2.hpp"
#include "glm/matrix.hpp"

struct Constants {
  glm::vec3 *position;
  glm::vec3 *velocity;
  glm::vec3 *force;
  glm::ivec4 *quads;
  float *A;
  glm::mat3 *DmInv;
};

__constant__ Constants cuConstants;

/* The data in the simulator must have been initialized, 
   i.e. createMesh() before setUp() */
void Solver::setUp(Simulator *sim) {
  this->sim = sim;
  int device_count = 0;
  std::string name;
  hipError_t err = hipGetDeviceCount(&device_count);

  printf("---------------------------------------------------------\n");
  printf("Initializing CUDA for CudaRenderer\n");
  printf("Found %d CUDA devices\n", device_count);

  for (int i = 0; i < device_count; i++) {
    hipDeviceProp_t deviceProps;
    hipGetDeviceProperties(&deviceProps, i);
    name = deviceProps.name;
    printf("Device %d: %s\n", i, deviceProps.name);
    printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
    printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
    printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
  }
  printf("---------------------------------------------------------\n");

  hipMalloc(&cudaDevicePosition,     N_POINTS * sizeof(glm::vec2));
  hipMalloc(&cudaDeviceVelocity,     N_POINTS * sizeof(glm::vec2));
  hipMalloc(&cudaDeviceForce, N_POINTS * sizeof(glm::vec2));
  hipMalloc(&cudaDeviceQuads,    N_QUADS * sizeof(glm::ivec3));
  cp2GPU();

  hipMalloc(&cudaDeviceA,            N_QUADS * sizeof(float));
  hipMalloc(&cudaDeviceDmInv,        N_QUADS * sizeof(glm::mat2));

  Constants constants;
  constants.position = cudaDevicePosition;
  constants.velocity = cudaDeviceVelocity;
  constants.force = cudaDeviceForce;
  constants.quads = cudaDeviceQuads;
  constants.DmInv = cudaDeviceDmInv;
  constants.A = cudaDeviceA;

  hipMemcpyToSymbol(HIP_SYMBOL(cuConstants), &constants, sizeof(Constants));

}

void Solver::cp2GPU() {
  hipMemcpy(cudaDevicePosition,      sim->x,     N_POINTS * sizeof(glm::vec2), hipMemcpyHostToDevice);
  hipMemcpy(cudaDeviceVelocity,      sim->v,     N_POINTS * sizeof(glm::vec2), hipMemcpyHostToDevice);
  hipMemcpy(cudaDeviceForce,         sim->f,     N_POINTS * sizeof(glm::vec2), hipMemcpyHostToDevice);
  hipMemcpy(cudaDeviceQuads, sim->quads, N_QUADS * sizeof(glm::ivec3), hipMemcpyHostToDevice);
}

void Solver::cp2CPU() {
  hipMemcpy(sim->x, cudaDevicePosition, N_POINTS * sizeof(glm::vec2), hipMemcpyDeviceToHost);
}

__global__
void kernelComputeDmInv() {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int a = cuConstants.quads[i].x;
  int b = cuConstants.quads[i].y;
  int c = cuConstants.quads[i].z;
  int d = cuConstants.quads[i].w;
  glm::mat3 Dm = glm::mat3(cuConstants.position[b] - cuConstants.position[a], cuConstants.position[c] - cuConstants.position[a], cuConstants.position[d] - cuConstants.position[a]);
  cuConstants.DmInv[i] = glm::inverse(Dm);
  cuConstants.A[i] = 0.5f * glm::abs(glm::determinant(Dm));
}

void Solver::computeDmInv() {
  kernelComputeDmInv<<<N_QUADS, 1>>>();
  hipDeviceSynchronize();
}

__global__
void kernelComputeForces() {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int a = cuConstants.quads[i].x;
  int b = cuConstants.quads[i].y;
  int c = cuConstants.quads[i].z;
  glm::mat3 Ds = glm::mat2(cuConstants.position[b] - cuConstants.position[a], cuConstants.position[c] - cuConstants.position[a]);
  glm::mat3 F = Ds * cuConstants.DmInv[i];
  glm::mat3 E = 0.5f * (glm::transpose(F) * F - glm::identity<glm::mat3>());
  glm::mat3 P = F * (2.0f * params::mu * E);
  glm::mat3 grad = glm::transpose(cuConstants.A[i] * P * glm::transpose(cuConstants.DmInv[i]));
  atomicAdd(&cuConstants.force[b].x, grad[0][0]);
  atomicAdd(&cuConstants.force[b].y, grad[1][0]);
  atomicAdd(&cuConstants.force[c].x, grad[0][1]);
  atomicAdd(&cuConstants.force[c].y, grad[1][1]);
  atomicAdd(&cuConstants.force[a].x, -grad[0][0] - grad[0][1]);
  atomicAdd(&cuConstants.force[a].y, -grad[1][0] - grad[1][1]);
}

__global__
void kernelUpdatePosition() {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  glm::vec3 acc = -cuConstants.force[i] - glm::vec3(0.0f, params::g, 0.0f);
  cuConstants.velocity[i] += acc * params::dt;
  cuConstants.position[i] += cuConstants.velocity[i] * params::dt;
  cuConstants.velocity[i] *= glm::exp(-params::dt * params::damping);
  cuConstants.force[i] = glm::vec3(0.0f, 0.0f, 0.0f);

  if (cuConstants.position[i].y < -1.0f) {
    cuConstants.position[i].y = -1.0f;
    cuConstants.velocity[i].y = 0.0f;
    cuConstants.velocity[i].x *= 0.9f;
  }
}

void Solver::solve() {
  for (int i = 0; i < params::sub_steps; i++) {
    kernelComputeForces<<<N_QUADS, 1>>>();
    hipDeviceSynchronize();
    kernelUpdatePosition<<<N_POINTS, 1>>>();
    hipDeviceSynchronize();
  }
  cp2CPU();
}

